
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <unistd.h>
#include <string>
#include <stdio.h>

using std::cout;
using std::endl;
using std::vector;
using std::ifstream;
using std::swap;
using std::string;
using namespace std;

#define ALIVE 'X'
#define DEAD '-'
#define THREADS 512

__global__
void play_game(int rows, int cols, char d_current_gen[], char d_next_gen[]) {

	int curr_cell = blockIdx.x * blockDim.x + threadIdx.x;

	if (curr_cell < rows * cols) {

		// Row and column indexes of current cell
		int row_idx = curr_cell / cols;
		int col_idx = curr_cell % cols;

		int curr_nbr;
		int nbr_row;
		int nbr_col;

		int num_alive = 0;

		// Loop through to find every neighbour for the current cell
		for (size_t i = row_idx - 1; i <= row_idx + 1; i++) {
			for (size_t j = col_idx - 1; j <= col_idx + 1; j++) {

				nbr_row = i;
				nbr_col = j;

				// Implementing world wrapping
				if (nbr_row < 0) {
					nbr_row += rows;
				}

				if (nbr_col < 0) {
					nbr_col += cols;
				}

				if (nbr_row == rows) {
					nbr_row = 0;
				}

				if (nbr_col == cols) {
					nbr_col = 0;
				}

				// Formula for calculating the current neighbour
				curr_nbr = nbr_row * cols + nbr_col;

				// Continue if the neighbour == the current cell
				if (curr_nbr == curr_cell) {
					continue;
				}

				// Increment count of ALIVE neighbours if the neighbour is ALIVE
				if (d_current_gen[curr_nbr] == ALIVE) {
					num_alive++;
				}
			}
		}

		// If curr_cell is ALIVE
		if (d_current_gen[curr_cell] == ALIVE) {

			// If num live neighbours is < 2 or > 3, kill it
			if (num_alive < 2 || num_alive > 3) {
				d_next_gen[curr_cell] = DEAD;

				// Else if num live neighbours == 2 || == 3
			} else {
				d_next_gen[curr_cell] = ALIVE;
			}

			// Else if curr_cell is DEAD
		} else {

			// If num live neighbours == 3, make it alive
			if (num_alive == 3) {
				d_next_gen[curr_cell] = ALIVE;
			} else {
				d_next_gen[curr_cell] = DEAD;
			}
		}
	}
}

// Function for printing a grid
void print_grid(int rows, int cols, char grid[]) {

	cout << "\n";

	for (size_t i = 0; i < rows; i++) {
		for (size_t j = 0; j < cols; j++) {
			cout << grid[i * cols + j];
		}
		cout << "\n";
	}
}

int main(int argc, char * argv[]) {

	int opt;
	int num_iterations = 1;
	bool verbose = false;

	string extension = ".txt";

	// Reject the run if no file specified
	if ((string(argv[argc - 1]).find(extension)) == std::string::npos) {
		cout << "Error: You must specify a .txt file as the last parameter (./cugol -i 100 -v input.txt)\n";
		return EXIT_FAILURE;
	}

	while ((opt = getopt(argc, argv, "i:v:")) != -1) {
		switch (opt) {
		case 'i':
			num_iterations = atoi(argv[2]);
			break;
		case 'v':
			verbose = true;
			break;
		}
	}

	int rows = 0;
	int cols = 0;

	// Temporary vector to store unknown amount of characters from input file
	vector<char> temp;

	char c;

	// Read in data to temp
	ifstream fn;

	fn.open(argv[argc - 1], ifstream::in);

	while (!fn.eof()) {

		// If a newline is reached, increment the number of rows
		if (fn.peek() == '\n' || fn.peek() == '\r') {
			rows++;
		}

		fn >> c;
		temp.push_back(c);
	}

	fn.close();

	// We need to increment rows again because it will not find a newline at the end of the file
	rows++;

	cols = temp.size() / rows;

	// Declare array and memory allocation size.
	int array_size = rows * cols;
	const int ARRAY_BYTES = sizeof(char) * array_size;

	// Host arrays and allocation of host memory
	char * h_current_gen = (char *) malloc(ARRAY_BYTES);
	char * h_next_gen = (char *) malloc(ARRAY_BYTES);

	// Copy data from vector to host array.
	for (size_t i = 0; i < array_size; i++) {
		h_current_gen[i] = temp[i];
	}

//	// Printing out the initial state of the game
	print_grid(rows, cols, h_current_gen);

	// Device arrays
	char * d_current_gen;
	char * d_next_gen;

	// Allocate GPU memory
	hipMalloc((void**) &d_current_gen, ARRAY_BYTES);
	hipMalloc((void**) &d_next_gen, ARRAY_BYTES);

	// Transfer memory to the GPU
	hipMemcpy(d_current_gen, h_current_gen, ARRAY_BYTES,
			hipMemcpyHostToDevice);

	// Loop for as many iterations as was specified (1 if unspecified)
	for (size_t i = 0; i < num_iterations; i++) {

		// Launch kernel
		play_game<<<(array_size + THREADS - 1) / THREADS, THREADS>>>(rows, cols,
				d_current_gen, d_next_gen);

		// Synchronise threads
		hipDeviceSynchronize();

		// Memcpy back to host and print the new generation if verbose was set
		if (verbose) {
			hipMemcpy(h_next_gen, d_next_gen, ARRAY_BYTES,
					hipMemcpyDeviceToHost);

			print_grid(rows, cols, h_next_gen);
		}

		// Pass the memory of d_next_gen over to d_current_gen
		swap(d_current_gen, d_next_gen);
	}

	// Transfer memory from GPU back to host
	hipMemcpy(h_next_gen, d_next_gen, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// Print final result of game only if verbose was not set
	if (!verbose) {
		print_grid(rows, cols, h_next_gen);
	}

	// Free CPU memory
	free(h_current_gen);
	free(h_next_gen);

	// Free GPU memory
	hipFree(d_current_gen);
	hipFree(d_next_gen);

	return EXIT_SUCCESS;
}
